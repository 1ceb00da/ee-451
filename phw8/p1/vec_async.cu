
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <time.h>

#define size 65536


__global__ void vector_add(int *a, int *b, int *c){
	int my_id;
	my_id = blockIdx.x*blockDim.x + threadIdx.x;	
	c[my_id] = a[my_id] + b[my_id];  
}

int main(){		
    int i;
    int *a ;
	int *b ;
    int *c ;
	
	hipHostMalloc((void**)&a, sizeof(int)*size, hipHostMallocDefault);	
	hipHostMalloc((void**)&b, sizeof(int)*size, hipHostMallocDefault);	
	hipHostMalloc((void**)&c, sizeof(int)*size, hipHostMallocDefault);	
	
	  for(i=0; i<size; i++){
			a[i]=100;
			b[i]=2;
  	}
		int *gpu_a, *gpu_b, *gpu_c;
		hipMalloc((void**)&gpu_a, sizeof(int)*size); 
		hipMalloc((void**)&gpu_b, sizeof(int)*size);
		hipMalloc((void**)&gpu_c, sizeof(int)*size);
			  
		
		dim3 dimGrid(32);
		dim3 dimBlock(1024);
		
		hipStream_t stream[2];
		hipStreamCreate (&stream[0]);
		hipStreamCreate (&stream[1]);
		
		hipMemcpyAsync(gpu_a, a, sizeof(int)*size/2, hipMemcpyHostToDevice, stream[0]);
		hipMemcpyAsync(gpu_b, b, sizeof(int)*size/2, hipMemcpyHostToDevice, stream[0]);
		vector_add<<<dimGrid, dimBlock, 0, stream[0]>>>(gpu_a, gpu_b, gpu_c);						
		hipMemcpyAsync(c, gpu_c, sizeof(int)*size/2, hipMemcpyDeviceToHost, stream[0]);
		
		hipMemcpyAsync(&gpu_a[size/2], &a[size/2], sizeof(int)*size/2, hipMemcpyHostToDevice, stream[1]);
		hipMemcpyAsync(&gpu_b[size/2], &b[size/2], sizeof(int)*size/2, hipMemcpyHostToDevice, stream[1]);
		vector_add<<<dimGrid, dimBlock, 0, stream[1]>>>(&gpu_a[size/2], &gpu_b[size/2], &gpu_c[size/2]);						
		hipMemcpyAsync(&c[size/2], &gpu_c[size/2], sizeof(int)*size/2, hipMemcpyDeviceToHost, stream[1]);
		
		hipStreamSynchronize(stream[0]);
		hipStreamSynchronize(stream[1]);
		hipStreamDestroy(stream[0]);
		hipStreamDestroy(stream[1]);
		
		for(i=size/2; i<size/2+12; i++)
			printf("c[%d]=%d ", i, c[i]);
  	
		hipFree(a);
		hipFree(b);
		hipFree(c);
		hipFree(gpu_a);  
		hipFree(gpu_b);  
		hipFree(gpu_c);  
		return 0;
}	
