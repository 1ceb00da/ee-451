
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define n 1024

__global__ void mul_mat(int *a, int *b, int *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;

	int local;


	for (i = 0; i < n; i++)
		local += a[myx+((n)*i)] * b[((n)*i)+myy];
	printf("%d \n", local);

	c[(myx*(n))+myy] = local;
}

int main() {
	int i;

	int *a;
	int *b;
	int *c;

	hipHostMalloc((void**)&a, sizeof(int)*n*n, hipHostMallocDefault);  
	hipHostMalloc((void**)&b, sizeof(int)*n*n, hipHostMallocDefault);  
	hipHostMalloc((void**)&c, sizeof(int)*n*n, hipHostMallocDefault);  

	hipEvent_t start, stop;
	float time;

	dim3 dimGrid(64,64);
	dim3 dimBlock(16,16);

	for(i=0;i<n*n;i++) {
		a[i] = 1;
		b[i] = 2;
		c[i] = 0;
	}

	int *gpua, *gpub, *gpuc;

	hipMalloc((void**)&gpua, sizeof(int)*n*n);
	hipMalloc((void**)&gpub, sizeof(int)*n*n);
	hipMalloc((void**)&gpuc, sizeof(int)*n*n);

	hipStream_t stream[2];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	
	hipMemcpyAsync(gpua, a, sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(gpub, b, sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[0]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[0]>>> (gpua, gpub, gpuc);
	hipMemcpyAsync(c, gpuc, sizeof(int)*n*n/2, hipMemcpyDeviceToHost, stream[0]);
	
	hipMemcpyAsync(&gpua[n*n/2], &a[n*n/2], sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(&gpub[n*n/2], &b[n*n/2], sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[1]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[1]>>> (&gpua[n*n/2], &gpub[n*n/2], &gpuc[n*n/2]);
	hipMemcpyAsync(&c[n*n/2], &gpuc[n*n/2], sizeof(int)*n*n/2, hipMemcpyDeviceToHost, stream[1]);

	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);
	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);

	printf("C[451][451] = %d\n",c[451*n + 451]);
	printf("Time - %f\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(a);
	hipFree(b);
	hipFree(c);

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	return 0;

}
