
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define n 1024

__global__ void mul_mat(int *a, int *b, int *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;

	int local;

	for (i = 0; i < n; i++)
		local += a[myx+n*i] * b[n*i+myy];

	c[myx*n+myy] = local;
}

int main() {
	int i;

	int *a;
	int *b;
	int *c;
	hipEvent_t start, stop;
	float time;

	dim3 dimGrid(64,64);
	dim3 dimBlock(16,16);

    hipHostMalloc((void**)&a, sizeof(int)*n*n, hipHostMallocDefault);	
	hipHostMalloc((void**)&b, sizeof(int)*n*n, hipHostMallocDefault);	
	hipHostMalloc((void**)&c, sizeof(int)*n*n, hipHostMallocDefault);	

	for(i=0;i<n*n;i++) {
		a[i] = 1;
		b[i] = 2;
		c[i] = 0;
	}

	int *gpua, *gpub, *gpuc;

	hipMalloc((void**)&gpua, sizeof(int)*n*n);
	hipMalloc((void**)&gpub, sizeof(int)*n*n);
	hipMalloc((void**)&gpuc, sizeof(int)*n*n);
	
    hipStream_t stream[4];
	hipStreamCreate (&stream[0]);
	hipStreamCreate (&stream[1]);
    hipStreamCreate (&stream[2]);
	hipStreamCreate (&stream[3]);

        
    hipEventCreate(&start);
	hipEventCreate(&stop);
	
    // start
    hipEventRecord(start,0);
    
    hipMemcpyAsync(gpua, a, sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(gpub, b, sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[0]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[0]>>> (gpua, gpub, gpuc);
	hipMemcpyAsync(c, gpuc, sizeof(int)*n*n/4, hipMemcpyDeviceToHost, stream[0]);
    
    hipMemcpyAsync(&gpua[n*n/4], &a[n*n/4], sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(&gpub[n*n/4], &b[n*n/4], sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[1]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[1]>>> (gpua, gpub, gpuc);
	hipMemcpyAsync(&c[n*n/4], &gpuc[n*n/4], sizeof(int)*n*n/4, hipMemcpyDeviceToHost, stream[1]);

    hipMemcpyAsync(&gpua[n*n/2], &a[n*n/2], sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[2]);
	hipMemcpyAsync(&gpub[n*n/2], &b[n*n/2], sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[2]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[2]>>> (gpua, gpub, gpuc);
	hipMemcpyAsync(&c[n*n/2], &gpuc[n*n/2], sizeof(int)*n*n/4, hipMemcpyDeviceToHost, stream[2]);

    hipMemcpyAsync(&gpua[3*n*n/4], &a[3*n*n/4], sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[2]);
	hipMemcpyAsync(&gpub[3*n*n/4], &b[3*n*n/4], sizeof(int)*n*n/4, hipMemcpyHostToDevice, stream[2]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[3]>>> (gpua, gpub, gpuc);
	hipMemcpyAsync(&c[3*n*n/4], &gpuc[3*n*n/4], sizeof(int)*n*n/4, hipMemcpyDeviceToHost, stream[2]);

	// stop 
    hipEventRecord(stop,0);

	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);
	hipStreamSynchronize(stream[2]);
	hipStreamSynchronize(stream[3]);

	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	hipStreamDestroy(stream[2]);
	hipStreamDestroy(stream[3]);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);


	printf("C[451][451] = %d\n",c[451*1024 + 451]);
	printf("Time - %f\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
    
    hipFree(a);
	hipFree(b);
	hipFree(c);
    hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	return 0;

}
