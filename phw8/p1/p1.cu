
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define n 1024

__global__ void mul_mat(int *a, int *b, int *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;

	int local;

	for (i = 0; i < n; i++)
		local += a[myx+n*i] * b[n*i+myy];

	c[myx*n+myy] = local;
}

int main() {
	int i;

	int *a = (int*)malloc(sizeof(int)*n*n);
	int *b = (int*)malloc(sizeof(int)*n*n);
	int *c = (int*)malloc(sizeof(int)*n*n);
	hipEvent_t start, stop;
	float time;

	dim3 dimGrid(64,64);
	dim3 dimBlock(16,16);

	for(i=0;i<n*n;i++) {
		a[i] = 1;
		b[i] = 2;
		c[i] = 0;
	}

	int *gpua, *gpub, *gpuc;

	hipMalloc((void**)&gpua, sizeof(int)*n*n);
	hipMalloc((void**)&gpub, sizeof(int)*n*n);
	hipMalloc((void**)&gpuc, sizeof(int)*n*n);

	hipStream_t stream[2];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);

	hipMemcpyAsync(gpua, a, sizeof(int)*n*n, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(gpub, b, sizeof(int)*n*n, hipMemcpyHostToDevice, stream[0]);
	//cudaMemcpy(gpua, a, sizeof(int)*n*n, cudaMemcpyHostToDevice);
	//cudaMemcpy(gpub, b, sizeof(int)*n*n, cudaMemcpyHostToDevice);
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	mul_mat<<<dimGrid, dimBlock, 0, stream[0]>>> (gpua, gpub, gpuc);
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	hipMemcpyAsync(c, gpuc, sizeof(int)*n*n, hipMemcpyDeviceToHost, stream[0]);
	//cudaMemcpy(c, gpuc, sizeof(int)*n*n, cudaMemcpyDeviceToHost);

	hipStreamSynchronize(stream[0]);
	hipStreamDestroy(stream[0]);

	printf("C[451][451] = %d\n",c[451*1024 + 451]);
	printf("Time - %f\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(a);
	free(b);
	free(c);

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	return 0;

}
