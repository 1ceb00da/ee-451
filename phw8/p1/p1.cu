#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define n 1024

__global__ void mul_mat(int *a, int *b, int *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;

	int local;

	for (i = 0; i < n; i++)
		local += a[myx+n*i] * b[n*i+myy];

	c[myx*n+myy] = local;
}

int main() {
	int i;

	int *a = (int*)malloc(sizeof(int)*n*n);
	int *b = (int*)malloc(sizeof(int)*n*n);
	int *c = (int*)malloc(sizeof(int)*n*n);
	hipEvent_t start, stop;
	float time;

	dim3 dimGrid(64,64);
	dim3 dimBlock(16,16);

	for(i=0;i<n*n;i++) {
		a[i] = 1;
		b[i] = 2;
		c[i] = 0;
	}

	int *gpua, *gpub, *gpuc;

	hipMalloc((void**)&gpua, sizeof(int)*n*n);
	hipMalloc((void**)&gpub, sizeof(int)*n*n);
	hipMalloc((void**)&gpuc, sizeof(int)*n*n);

	hipStream_t stream[2];
	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	
	hipMemcpyAsync(gpua, a, sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(gpub, b, sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[0]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[0]>>> (gpua, gpub, gpuc);
	hipMemcpyAsync(c, gpuc, sizeof(int)*n*n/2, hipMemcpyDeviceToHost, stream[0]);
	
	hipMemcpyAsync(&gpua[n*n/2], &a[n*n/2], sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[1]);
	hipMemcpyAsync(&gpub[n*n/2], &b[n*n/2], sizeof(int)*n*n/2, hipMemcpyHostToDevice, stream[1]);
	mul_mat<<<dimGrid, dimBlock, 0, stream[1]>>> (&gpua[n*n/2], &gpub[n*n/2], &gpuc[n*n/2]);
	hipMemcpyAsync(&c[n*n/2], gpuc[n*n/2], sizeof(int)*n*n/2, hipMemcpyDeviceToHost, stream[1]);

	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	hipStreamSynchronize(stream[0]);
	hipStreamSynchronize(stream[1]);
	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);

	printf("C[451][451] = %d\n",c[451*1024 + 451]);
	printf("Time - %f\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(a);
	free(b);
	free(c);

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	return 0;

}
