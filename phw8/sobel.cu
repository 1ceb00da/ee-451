#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include<math.h>

#define n 192

#define input_file  "phw8.raw"
#define output_file "output.raw"

__global__ void sobel(unsigned char *a, unsigned char *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;

    
    unsigned char gx00=-1, gx01=-2, gx02=-1;
    unsigned char gx10=0, gx11=0, gx12=0;
    unsigned char gx20=1, gx21=2, gx22=1;
    
    unsigned char gy00=-1, gy01=0, gy02=1;
    unsigned char gy10=-2, gy11=0, gy12=2;
    unsigned char gy20=-1, gy21=0, gy22=1;
    
    
    unsigned char gx[3][3] =
    {
        {-1,2,-1},
        {0,0,0},
        {1,2,1}
    };
    
    unsigned char gy[3][3] =
    {
        {-1,0,1},
        {-2,0,2},
        {-1,0,1}
    };
    
    unsigned char dx = 0, dy = 0;

    dx += (a[((myx-1)*(n+2)) + myy-1] * gx[0][0])
       +  (a[((myx-1)*(n+2)) + myy] * gx[0][1])
       +  (a[((myx-1)*(n+2)) + myy+1] * gx[0][2])
       
       +  (a[((myx)*(n+2)) + myy-1] * gx[1][0])
       +  (a[((myx)*(n+2)) + myy] * gx[1][1])
       +  (a[((myx)*(n+2)) + myy+1] * gx[1][2])
       
       +  (a[((myx+1)*(n+2)) + myy-1] * gx[2][0])
       +  (a[((myx+1)*(n+2)) + myy] * gx[2][1])
       +  (a[((myx+1)*(n+2)) + myy+1] * gx[2][2]);
    
    dy += (a[((myx-1)*(n+2)) + myy-1] * gy[0][0])
       +  (a[((myx-1)*(n+2)) + myy] * gy[0][1])
       +  (a[((myx-1)*(n+2)) + myy+1] * gy[0][2])
       
       +  (a[((myx)*(n+2)) + myy-1] * gy[1][0])
       +  (a[((myx)*(n+2)) + myy] * gy[1][1])
       +  (a[((myx)*(n+2)) + myy+1] * gy[1][2])
       
       +  (a[((myx+1)*(n+2)) + myy-1] * gy[2][0])
       +  (a[((myx+1)*(n+2)) + myy] * gy[2][1])
       +  (a[((myx+1)*(n+2)) + myy+1] * gy[2][2]);
    
    
	c[myx*n+myy] = (unsigned char)sqrt( ( ((double)dx)*((double)dx) + ((double)dy)*((double)dy) ) );
}

int main() {
	unsigned char *a = (unsigned char*)malloc(sizeof(unsigned char)*(n+2)*(n+2));
	unsigned char *c = (unsigned char*)malloc(sizeof(unsigned char)*n*n);
    
	hipEvent_t start, stop;
	float time;
    
    FILE *fp;
	
    int numBlocks = 6;
    dim3 threadsPerBlock(32,32);
    
    
    // Read image
    if (!(fp=fopen(input_file, "rb"))) {
		printf("can not opern file\n");
		return 1;
	}
	fread(a, sizeof(unsigned char), (n+2)*(n+2), fp);
	fclose(fp);


	unsigned char *gpua, *gpuc;

	hipMalloc((void**)&gpua, sizeof(unsigned char *)*(n+2)*(n+2));
	hipMalloc((void**)&gpuc, sizeof(unsigned char *)*n*n);

	hipMemcpy(gpua, a, sizeof(unsigned char *)*(n+2)*(n+2), hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	sobel<<<numBlocks, threadsPerBlock>>> (gpua, gpuc);
	hipEventRecord(stop,0);


	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(c, gpuc, sizeof(unsigned char *)*n*n, hipMemcpyDeviceToHost);

    if (!(fp=fopen(output_file,"wb"))) {
		printf("can not opern file\n");
		return 1;
	}	
	fwrite(c, sizeof(unsigned char),n*n, fp);
    fclose(fp);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(a);
	free(b);
	free(c);

	hipFree(gpua);
	hipFree(gpuc);

	return 0;

}
