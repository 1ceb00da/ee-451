
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <limits.h>

#define TRUE 0
#define FALSE 1


typedef struct {
	
	int src;
	int dst;
	int cost;
} Edge;


__global__ void bellman_ford_kernel(int *dis_arr, Edge *edges, int *change) {
	int my_id;
	my_id = blockIdx.x*blockDim.x + threadIdx.x;

	Edge my_edge = edges[my_id];
	int curr_dis, cand_dis;


	int i;
	// test code
	for (i =0; i < 5; i++) {
		printf("%d->%d: %d\n", edges[i].src, edges[i].dst, edges[i].cost);
	}

	curr_dis = dis_arr[my_edge.dst];
	cand_dis = dis_arr[my_edge.src] + my_edge.cost;

	if (cand_dis < curr_dis) {
		*change = TRUE;
		dis_arr[my_edge.dst] = cand_dis;
	}
}

int main() {
	
	int *dis_arr;
	Edge *edges;

	int n = 5; // num vertices
	int m = 5; // num edges

	dis_arr = (int*)malloc(sizeof(int)*n);
	edges = (Edge*)malloc(sizeof(Edge)*m);

	int i;

	// init edges
	Edge e1 = {0,1, 5};
	Edge e2 = {0,3, 1};
	Edge e3 = {1,2, 3};
	Edge e4 = {3,4, 4};
	Edge e5 = {2,4, 8};
	edges[0] = e1;
	edges[1] = e2;
	edges[2] = e3;
	edges[3] = e4;
	edges[4] = e5;

	// init dis_arr
	// set each val to INT_MAX
	for (i=0; i < n; i++)
		dis_arr[i] = INT_MAX;

	// set source dis
	dis_arr[0] = 0;

	int *gpu_dis_arr;
	Edge *gpu_edges;

	hipMalloc((void**)&gpu_dis_arr, sizeof(int)*n);
	hipMalloc((void**)&gpu_edges, sizeof(Edge)*m);

	hipMemcpy(gpu_dis_arr, dis_arr, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(gpu_edges, edges, sizeof(Edge)*m, hipMemcpyHostToDevice);

	int *change;
	int *gpu_change;

	change = (int*)malloc(sizeof(int));
	hipMalloc((void**)&gpu_change, sizeof(int));

	*change = TRUE;

	while(change == TRUE) {
		//*change = FALSE;
		hipMemcpy(gpu_change, change, sizeof(int)*1, hipMemcpyHostToDevice);
		bellman_ford_kernel<<< 1 , 5 >>>(gpu_dis_arr, gpu_edges, gpu_change);
		hipMemcpy(change, gpu_change, sizeof(int)*1, hipMemcpyDeviceToHost);
	}

	hipMemcpy(dis_arr, gpu_dis_arr, sizeof(int)*n, hipMemcpyDeviceToHost);

	// print dis-arr
	printf("Distance array\n");
	for (i=0; i < n; i++)
		printf("%d ", dis_arr[i]);

}



