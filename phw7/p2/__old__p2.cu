
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define n 1024

__global__ void mul_mat(int *a, int *b, int *c) {
	int myx, myy, i;
	myx = blockIdx.x * blockDim.x + threadIdx.x;
	myy = blockIdx.y * blockDim.y + threadIdx.y;

	int local;

	for (i = 0; i < n; i++)
		local += a[myx+n*i] * b[n*i+myy];

	c[myx*n+myy] = local;
}

int main() {
	int i;

	int *a = (int*)malloc(sizeof(int)*n*n);
	int *b = (int*)malloc(sizeof(int)*n*n);
	int *c = (int*)malloc(sizeof(int)*n*n);
	hipEvent_t start, stop;
	float time;

	dim3 dimGrid(32,32);
	dim3 dimBlock(32,32);

	for(i=0;i<n*n;i++) {
		a[i] = 1;
		b[i] = 2;
		c[i] = 0;
	}

	int *gpua, *gpub, *gpuc;

	hipMalloc((void**)&gpua, sizeof(int)*n*n);
	hipMalloc((void**)&gpub, sizeof(int)*n*n);
	hipMalloc((void**)&gpuc, sizeof(int)*n*n);

	hipMemcpy(gpua, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(gpub, b, sizeof(int)*n*n, hipMemcpyHostToDevice);
	

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);	
	mul_mat<<<dimGrid, dimBlock>>> (gpua, gpub, gpuc);
	hipEventRecord(stop, 0);
	
	hipEventSynchronize(stop);

	hipMemcpy(c, gpuc, sizeof(int)*n*n, hipMemcpyDeviceToHost);
	
	hipEventElapsedTime(&time, start, stop);
	printf("C[451][451] = %d\n",c[451*1024 + 451]);
	printf("Time - %f\n", time);

	free(a);
	free(b);
	free(c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	return 0;

}
