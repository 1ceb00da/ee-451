
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define n 1024
#define block_size 32

__global__ void mul_mat(int *a, int *b, int *c) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	int row = threadIdx.y;
	int col = threadIdx.x;

	int i,j;

	int myx = blockIdx.x * blockDim.x + threadIdx.x;
	int myy = blockIdx.y * blockDim.y + threadIdx.y;

	int local=0;
	
	__shared__ int As[32][32];
	__shared__ int Bs[32][32];
	

	for(i=0;i<n/block_size;i++) {
		As[row][col] = a[myx*n + (i*blockDim.y + col)];
		Bs[row][col] = b[(i*blockDim.x+row)*n + myy];
		__syncthreads();

		for(j=0;j<block_size;j++)
			local += As[row][j]*Bs[j][col];
		__syncthreads();
	}

	c[myx*n+myy] = local;
}

int main() {
	int i;

	int *a = (int*)malloc(sizeof(int)*n*n);
	int *b = (int*)malloc(sizeof(int)*n*n);
	int *c = (int*)malloc(sizeof(int)*n*n);
	hipEvent_t start, stop;
	float time;

	dim3 dimGrid(32,32);
	dim3 dimBlock(32,32);

	for(i=0;i<n*n;i++) {
		a[i] = 1;
		b[i] = 2;
		c[i] = 0;
	}

	int *gpua, *gpub, *gpuc;

	hipMalloc((void**)&gpua, sizeof(int)*n*n);
	hipMalloc((void**)&gpub, sizeof(int)*n*n);
	hipMalloc((void**)&gpuc, sizeof(int)*n*n);

	hipMemcpy(gpua, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
	hipMemcpy(gpub, b, sizeof(int)*n*n, hipMemcpyHostToDevice);
	

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);	
	mul_mat<<<dimGrid, dimBlock>>> (gpua, gpub, gpuc);
	hipEventRecord(stop, 0);
	
	hipEventSynchronize(stop);

	hipMemcpy(c, gpuc, sizeof(int)*n*n, hipMemcpyDeviceToHost);
	
	hipEventElapsedTime(&time, start, stop);
	printf("C[451][451] = %d\n",c[451*1024 + 451]);
	printf("Time - %f\n", time);

	free(a);
	free(b);
	free(c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(gpua);
	hipFree(gpub);
	hipFree(gpuc);

	return 0;

}
